#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctime>
#include <hip/hip_runtime.h>

void allocate_matrix(int **matrix, int N){
    *matrix = (int*) malloc(sizeof(int) * N * N); 
}
void allocate_matrix(float **matrix, int N){
    *matrix = (float*) malloc(sizeof(float) * N * N); 
}
void generate_matrix(int *matrix, int N){
    for(int i = 0; i < N; i++){
       for(int j = 0; j < N; j++){
           *(matrix + i*N +j) = rand() % N + 10;
       } 
    }
}
void output_matrix(int *matrix, int N) {
    for(int i = 0; i < N; i++){
       printf("\n");
       for(int j = 0; j < N; j++){
           printf("%6d ",*(matrix + i*N +j));
       } 
    }
    printf("\n");
}
void output_matrix(float *matrix, int N) {
    for(int i = 0; i < N; i++){
       printf("\n");
       for(int j = 0; j < N; j++){
           printf("%6.2f ",*(matrix + i*N +j));
       } 
    }
    printf("\n");
}


texture<int, 1, hipReadModeElementType> matrix_t;

__global__ void LU(float *lu_matrix, const int N, bool * flag){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    if(id >=N) return; 
 
    for(int j = id; j < N; j+=blockDim.x*gridDim.x){
        if(tex1Dfetch(matrix_t, j) == 0) *flag = false;
        *(lu_matrix+j) = (float) tex1Dfetch(matrix_t, j);
    }
 
    __syncthreads();
    if(!*flag) return;
 
    for(int r = 1; r < N; r++){
        int temp_id = id;
        while(temp_id < r){
          temp_id+=blockDim.x*gridDim.x;
        }
        if(temp_id >= N) return ;
        __syncthreads();
        

        //L matrix calculation
        for(int i = temp_id; i < N; i+=blockDim.x*gridDim.x){
            *(lu_matrix+i*N + r-1) = (float) tex1Dfetch(matrix_t, i*N+r-1);
            for(int k = 0; k < r-1; k++)
            {
                *(lu_matrix+(i)*N+r-1) -= (*(lu_matrix+i*N+k)) * (*(lu_matrix+k*N + r-1));
            }
            
            *(lu_matrix+(i)*N+r-1) /= *(lu_matrix+(r-1)*N+r-1);
        }

        __syncthreads();

        //U matrix calculation
        for(int j = temp_id; j < N; j+=blockDim.x*gridDim.x){
            *(lu_matrix+r*N+j) = (float) tex1Dfetch(matrix_t, r*N+j);
            
            for(int k = 0; k <r; k++){
                *(lu_matrix+r*N+j) -= (*(lu_matrix+r*N+k)) * (*(lu_matrix+k*N + j));
            }
            if(tex1Dfetch(matrix_t, r*N+j) == 0)  *flag = false;
        }
        __syncthreads();
        if(!*flag) return;
    }
}

int gpu_solution(int *matrix, int N, float *lu_matrix){
    int * dev_matrix, threads;
    float * dev_lu_matrix, elapsed_time;
	  hipEvent_t start, stop;
    bool flag = true, * dev_flag;

    hipEventCreate(&start);
	  hipEventCreate(&stop);
 
	  hipMalloc(&dev_flag, sizeof(bool));
  	hipMalloc(&dev_matrix, sizeof(int) * N * N);
	  hipMalloc(&dev_lu_matrix, sizeof(float) * N * N);
 
    hipMemcpy(dev_matrix, matrix, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_flag, &flag, sizeof(bool), hipMemcpyHostToDevice);
    
 
    hipBindTexture(NULL, matrix_t, dev_matrix, hipCreateChannelDesc<int>(), sizeof(int) * N * N);
    
    if(N < 250){
        threads = N;
    }
    else{
        threads = 250;
    }

    hipEventRecord(start, 0);
 
  	LU<<<1,threads>>>(dev_lu_matrix, N, dev_flag);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	  
    hipEventElapsedTime(&elapsed_time, start, stop);
	  
    hipEventDestroy(start);
  	hipEventDestroy(stop);
    
    hipMemcpy(&flag, dev_flag, sizeof(bool), hipMemcpyDeviceToHost);
    if(flag){
      hipMemcpy(lu_matrix, dev_lu_matrix, sizeof(float) * N * N, hipMemcpyDeviceToHost);
    }
	  hipFree(dev_matrix);
	  hipFree(dev_lu_matrix);
 
    if(!flag) return -1;
 
	  return round(elapsed_time);
}

int cpu_solution(int *matrix, int N, float *lu_matrix){
    clock_t start = clock();
    
    for(int j = 0; j < N; j++){
        if((*(matrix+j)) == 0) return -1;
        *(lu_matrix+j) = (float) *(matrix+j);
    }
    
    for(int r = 1; r < N; r++){
        for(int i = r; i < N; i++)
        {
  
            *(lu_matrix+i*N + r-1) = (float) *(matrix+i*N+r-1);
            for(int k = 0; k < r-1; k++){
                *(lu_matrix+(i)*N+r-1) -= (*(lu_matrix+i*N+k)) * (*(lu_matrix+k*N + r-1));
            }
            
            *(lu_matrix+(i)*N+r-1) /= *(lu_matrix+(r-1)*N+r-1);         
        }
        

        for(int j = r; j < N; j++){
            *(lu_matrix+r*N+j) = (float) *(matrix+r*N+j);
            
            for(int k = 0; k <r; k++){
                *(lu_matrix+r*N+j) -= (*(lu_matrix+r*N+k)) * (*(lu_matrix+k*N + j));
            }
            
            if((*(matrix+(r)*N+j)) == 0) 
                return -1;
        }
    }
    
    return (int)(clock() - start)/ (CLOCKS_PER_SEC / 1000);
}



int main() {
    int N=64;
   /* int test_matrix[16] = {
        3, 4, -9, 5,
        -15, -12, 50, -16,
        -27, -36, 73, 8,
        9, 12, -10, -16
    };
*/
    int * matrix;
    float * cpu_lu_matrix, * gpu_lu_matrix;
    int time_cpu, time_gpu;
  /*  
    matrix = test_matrix;
    
    printf("\nИсходная матрица:\n");
    output_matrix(matrix, N);
    
    allocate_matrix(&cpu_lu_matrix, N);
    
    time_cpu = cpu_solution(matrix, N, cpu_lu_matrix);
    
  
    allocate_matrix(&gpu_lu_matrix, N);
    time_gpu = gpu_solution(matrix, N, gpu_lu_matrix);
 
  
    printf("\nLU GPU:\n");
    output_matrix(gpu_lu_matrix, N);

    free(gpu_lu_matrix);
 
    if(time_cpu >= 0 && time_gpu >= 0){
        printf("\nTime on CPU - %d ms\nTime on GPU - %d ms\n",time_cpu, time_gpu);
    }
*/
//N-matrix
    allocate_matrix(&matrix, N);
    generate_matrix(matrix, N);

    allocate_matrix(&cpu_lu_matrix, N);
    time_cpu = cpu_solution(matrix, N, cpu_lu_matrix);
    
    allocate_matrix(&gpu_lu_matrix, N);
    time_gpu = gpu_solution(matrix, N, gpu_lu_matrix);

 
    if(time_cpu >= 0 && time_gpu >= 0){
        printf("\nTime on CPU - %d ms\nTime on GPU - %d ms\n",time_cpu, time_gpu);
    }

    srand(time(NULL));
    return 0;
}

